
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 16

void random_ints(int *a, int n)
{
    for (int i = 0; i < n; i++)
    {
        a[i] = rand() % 10;
    }
}

__global__ void add(int *a, int *b, int *c)
{
    c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

int main()
{
    int size = sizeof(int) * N;
    int *a = (int *)malloc(size);
    random_ints(a, N);
    int *b = (int *)malloc(size);
    random_ints(b, N);
    int *c = (int *)malloc(size);
    memset(c, 0, size);


    // device copies of a, b, c
    int *d_a, *d_b, *d_c;

    // Allocate memory to device
    // Allocate memory to d_b and d_c using the following example
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);


    //// Copy inputs to device
    //// Copy input to d_b using the following example
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    //// Launch kernel
    add<<<N, 1>>>(d_a, d_b, d_c);
    hipDeviceSynchronize();

    //// Copy result from device to host
    //// What is the difference of this line compared to above cudaMemcpy?
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    //// print result, is result correct?
    for (int i = 0; i < N; i++)
    {
        printf("i = %d, %d + %d = %d\n", i, a[i], b[i], c[i]);
    }

    //// Memory cleanup!
    //// Clean up memory for d_a, d_b, d_c with following example.
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    //// Of course the same for host memory, too.
    free(a);
    free(b);
    free(c);

    return 0;
}
