
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 16

void random_ints(int *a, int n)
{
    for (int i = 0; i < n; i++)
    {
        a[i] = rand() % 10;
    }
}

__global__ void add(int *a, int *b, int *c)
{
    //// We are using the same number of blocks as the array length
    //// Use blockIdx.x for this section
    c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

int main()
{
    int size = sizeof(int) * N;
    int *a = (int *)malloc(size);
    random_ints(a, N);
    int *b = (int *)malloc(size);
    random_ints(b, N);
    int *c = (int *)malloc(size);
    memset(c, 0, size);

    // device copies of a, b, c
    int *d_a, *d_b, *d_c;

    // Allocate memory to d_a, d_b, and d_c
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    //// Copy a, b to d_a, d_b
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    //// Launch kernel
    //// N is placed in left side of the brackets (blocks)
    add<<<N, 1>>>(d_a, d_b, d_c);
    hipDeviceSynchronize();

    //// Copy result from device (d_c) to host (c)
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    //// print result, is result correct?
    for (int i = 0; i < N; i++)
    {
        printf("i = %d, %d + %d = %d\n", i, a[i], b[i], c[i]);
    }

    //// Memory cleanup!
    //// Clean up memory for d_a, d_b, d_c
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    //// Of course, we need to do the same for host memory
    free(a);
    free(b);
    free(c);

    return 0;
}
