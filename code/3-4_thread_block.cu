
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 32

#define BLOCK_SIZE 16

void random_ints(int *a, int n)
{
    for (int i = 0; i < n; i++)
    {
        a[i] = rand() % 10;
    }
}

__global__ void add(int *a, int *b, int *c)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    c[idx] = a[idx] + b[idx];
}

int main()
{
    int size = sizeof(int) * N;
    int *a = (int *)malloc(size);
    random_ints(a, N);
    int *b = (int *)malloc(size);
    random_ints(b, N);
    int *c = (int *)malloc(size);
    memset(c, 0, size);

    // device copies of a, b, c
    int *d_a, *d_b, *d_c;

    //// Allocate memory to d_a, d_b, and d_c
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);


    //// Copy input to d_a, d_b
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    //// Launch kernel
    int NUMBLOCKS = N / BLOCK_SIZE;
    add<<<NUMBLOCKS, BLOCK_SIZE>>>(d_a, d_b, d_c);
    hipDeviceSynchronize();

    //// Copy result from device (d_c) to host (c)
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    //// print result, is result correct?
    for (int i = 0; i < N; i++)
    {
        printf("i = %d, %d + %d = %d\n", i, a[i], b[i], c[i]);
    }

    //// Memory cleanup!
    //// Clean up memory for d_a, d_b, d_c
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    //// Of course the same for host memory, too.
    free(a);
    free(b);
    free(c);

    return 0;
}
